#include <iostream>
#include <fstream>
#include <memory>
#include <cstdint>
#include <filesystem>
#include <string>
#include <hip/hip_runtime.h>

#define PADDED(i) (i + (i / 32))
#define WARP_SIZE 32

__global__ void computeHistogramKernel(const int* input, int* global_histogram, int N, int B) {
    extern __shared__ int shared_array[];

    int tid = threadIdx.x;
    int global_id = blockIdx.x * blockDim.x + tid;

    int warp_id = tid / WARP_SIZE;
    int lane_id = tid % WARP_SIZE;
    int warps_per_block = blockDim.x / WARP_SIZE;

    int* shared_hist = &shared_array[warp_id * B];

    for (int i = lane_id; i < B; i += WARP_SIZE) {
        shared_hist[i] = 0;
    }
    __syncthreads();

    if (global_id < N) {
        int bin = input[global_id];
        if (bin >= 0 && bin < B) {
            atomicAdd(&shared_hist[bin], 1);
        }
    }
    __syncthreads();

    if (tid < B) {
        int total = 0;
        for (int w = 0; w < warps_per_block; ++w) {
            total += shared_hist[w * B + tid];
        }
        atomicAdd(&global_histogram[tid], total);
    }
}


namespace solution {
    std::string compute(const std::string &input_path, int N, int B) {
        std::string sol_path = std::filesystem::temp_directory_path() / "student_histogram.dat";
        std::ofstream sol_fs(sol_path, std::ios::binary);
        std::ifstream input_fs(input_path, std::ios::binary);

        // Read input data on host
        const auto input_data = std::make_unique<int[]>(N);
        input_fs.read(reinterpret_cast<char*>(input_data.get()), sizeof(int) * N);
        input_fs.close();

        // Allocate and initialize histogram on host
        auto histogram = std::make_unique<int[]>(B);
        for (int i = 0; i < B; i++) histogram[i] = 0;

        // Allocate device memory
        int *d_input = nullptr;
        int *d_histogram = nullptr;
        hipMalloc(&d_input, sizeof(int) * N);
        hipMalloc(&d_histogram, sizeof(int) * B);

        // Copy input data to device
        hipMemcpy(d_input, input_data.get(), sizeof(int) * N, hipMemcpyHostToDevice);
        hipMemset(d_histogram, 0, sizeof(int) * B);

        // Kernel launch parameters
        int threads_per_block = 256;
        int blocks = (N + threads_per_block - 1) / threads_per_block;
        int warps_per_block = threads_per_block / WARP_SIZE;
        size_t shared_mem_size = warps_per_block * PADDED(B) * sizeof(int);

        // Launch naive kernel
        computeHistogramKernel<<<blocks, threads_per_block, shared_mem_size>>>(d_input, d_histogram, N, B);
        hipDeviceSynchronize();  // Ensure kernel is done

        // Copy result back to host
        hipMemcpy(histogram.get(), d_histogram, sizeof(int) * B, hipMemcpyDeviceToHost);

        // Cleanup
        hipFree(d_input);
        hipFree(d_histogram);

        // Write output
        sol_fs.write(reinterpret_cast<const char*>(histogram.get()), sizeof(int) * B);
        sol_fs.close();

        return sol_path;
    }
}
